
#include <hip/hip_runtime.h>
#include<iostream>
#include<algorithm>
#include<stdio.h>
#include<fstream>
#include <stdlib.h>
using namespace std;
#define REPEAT 1
#define Real double
#define STRIDE 1
#define CACHELINE 8
__global__ void VecAdd(Real* A, int* N, Real* d_time);


int main(int argc, char* argv[])
{
   if(argc != 2)
   {
     std::cout << "Wrong number of argument!! Exiting program !!!";
     return 0;
   }
      
  // struct timeval tv1, tv2;
   int N = atoi(argv[1]);
   //register long elapsed=0;
   Real *A, *d_A;
   int *d_N;// *B, *C, *d_A, *d_B, *d_C ;           
   std::ofstream fp;
   fp.open("/home/hpc/ihpc/ihpc002h/gpu-exp/Master-thesis/data/result.txt", std::ofstream::app);
    double *d_time, h_time;
    A  = (Real*)malloc(N*sizeof(Real)); 
   // B  = (Real*)malloc(N*sizeof(Real)); 
   // C  = (Real*)malloc(N*sizeof(Real));
    
    
    hipMalloc(&d_A, N*sizeof(Real));
    hipMalloc(&d_time, sizeof(double));
    hipMalloc(&d_N, sizeof(double));
   // cudaMalloc(&d_B, N*sizeof(Real));
   // cudaMalloc(&d_C, N*sizeof(Real));
    
    
    for(int i=0; i < N ; i++)
    {
      A[i] = (i+STRIDE)%N;
     // B[i] = rand()%5;
      //C[i] = 0.0;
    }
    
    hipMemcpy(d_A, A, N*sizeof(Real), hipMemcpyHostToDevice );
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice );
    //cudaMemcpy(d_C, C, N*sizeof(Real), cudaMemcpyHostToDevice );
    
    //for(int i=0 ; i < REPEAT ; i++)
    //{
    //gettimeofday(&tv1, NULL);                  
    VecAdd<<<1, 1>>>(d_A, d_N, d_time);
    //gettimeofday(&tv2, NULL);
    //elapsed += ((tv2.tv_sec-tv1.tv_sec)*1000000.0 + (tv2.tv_usec-tv1.tv_usec));
    //}
    
    hipMemcpy(&h_time, d_time, sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
   //cout << N << std::endl;
    
    //double td = (N * 8.0)/1024;
    //cout << h_time << " " << CLOCKS_PER_SEC << std::endl;
    
    fp << N*8.0/1024.0 << " " << h_time << std::endl;
    
    for(int i =0; i < N ; i++)
    {
     // cout << A[i] << std::endl;
    }
     //fs << std::endl;
   
    hipFree(d_A);
    //cudaFree(d_B);
    //cudaFree(d_C);
    fp.close();
}
